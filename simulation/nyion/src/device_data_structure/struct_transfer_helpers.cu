
#include <hip/hip_runtime.h>


template <class TYPE>
void constructificate_array_on_struct(TYPE ** array, uint32_t N){
    TYPE * device_temp;         \
    /* malloc an array on the device, */ \
    gpu_error_check(hipMalloc(&device_temp, (N)*sizeof(* device_temp))); \
    /* memset the whole array, not just a buffer_end_pointer's worth */ \
    gpu_error_check(hipMemset(device_temp,0,(N)*sizeof(* device_temp))); \
    /* then copy the pointer to the new array into the struct. */ \
    gpu_error_check(hipMemcpy(array, &device_temp, sizeof(array), hipMemcpyHostToDevice));
}


template <class TYPE>
void defenestrate_array_from_struct(TYPE ** array){
    TYPE * temp;
    /* //copy the pointer to the data */
    gpu_error_check(hipMemcpy(&temp, array, sizeof(array), hipMemcpyDeviceToHost));
    /* //then destroy it! */
    gpu_error_check(hipFree(temp));
}
