#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"
#include "host_data_structure.hpp"

//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!

__global__ void test_fill(physics_mesh &d_a) {
    for(int i = 0; i < 10; i++){
        d_a.potential[i] = i;
    }
}



__host__ void CUDA_struct_copy_test(){
    //cuda Unified Memory with ManagedMemcpy would be much simpler,
    //but I think I prefer to see precisely when a PCIe transfer will occur.


    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    //input, which we don't want to mess with
    physics_mesh * host_input = new physics_mesh(mesh_sizes, 1);
    //so we create a surrogate to copy values to, without destructors
    physics_mesh * host_b = new physics_mesh(1);
    physics_mesh * device_a;

    host_input->potential[10] = 100;
    //copy all the physics arrays
    POTENTIAL_TYPE * device_potential;
    gpu_error_check(hipMalloc(&device_potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE)));
                        //order is (device, host
    gpu_error_check(hipMemcpy(device_potential, host_input->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyHostToDevice));

    //this step has to be done before the struct is copied to the device
    host_b->potential = device_potential;

    //copy the struct itself (and accessory array values on the stack)
    gpu_error_check(hipMalloc(&device_a, sizeof(physics_mesh)));
    gpu_error_check(hipMemcpy(device_a, host_b, sizeof(physics_mesh), hipMemcpyHostToDevice));

    test_fill<<<1, 1>>>(*device_a);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );


    // //and back again
    // physics_mesh * host_output = new physics_mesh(1); // no destructors!
    // gpu_error_check(hipMemcpy(host_output, device_a, sizeof(physics_mesh), hipMemcpyDeviceToHost));
    POTENTIAL_TYPE * output_potential;
    // host_output->potential =
    gpu_error_check(hipMemcpy(output_potential, device_a->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyDeviceToHost));


    hipFree(device_potential);
    hipFree(device_a->potential);
    hipFree(device_a);
    delete host_b;
    delete host_input;

    // delete [] host_output->potential;
    // delete host_output;
    // delete [] device_a->potential;
    delete output_potential;

}
