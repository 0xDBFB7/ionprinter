#include "device_data_structure.hpp"
#include "host_data_structure.hpp"


#include "gtest/gtest.h"

//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!



__host__ void CUDA_physics_mesh_copy_test(){
//     //cuda Unified Memory with ManagedMemcpy would be much simpler,
//     //but I think I prefer to see precisely when a PCIe transfer will occur.
//
//     int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
//     //input, which we don't want to mess with
//     physics_mesh * host_input = new physics_mesh(mesh_sizes, 1);
//     //so we create a surrogate to copy values to, without destructors
//     physics_mesh * host_b = new physics_mesh(1);
//     physics_mesh * device_a;

}
