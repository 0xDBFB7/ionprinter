#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"
#include "host_data_structure.hpp"


#include "gtest/gtest.h"

//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!

__global__ void test_fill(physics_mesh &d_a) {
    for(int i = 0; i < 10; i++){
        d_a.potential[i] = i;
    }
}

__host__ void CUDA_simple_copy_test(){
    const int N = 10;

    float * x = new float[N];
    for(int i = 0; i < N; i++){ x[i]= i;};
    float * d_x;

    gpu_error_check(hipMalloc(&d_x, N*sizeof(float)));
                    //always dest, src
    gpu_error_check(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
    gpu_error_check(hipDeviceSynchronize());

    gpu_error_check(hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost));
    gpu_error_check( hipDeviceSynchronize() );

    pretty_print_array(x, 0, N);

    gpu_error_check(hipFree(d_x));

    delete [] x;
}

struct test_struct{
    int test_int[10];
    float * storage;
    uint8_t * small_storage;
    //must try different size types
};

__global__ void test_fill_simple(test_struct * d_a) {
    for(int i = 0; i < 10; i++){
        d_a->storage[i] = d_a->test_int[5];
    }
    d_a->test_int[5] = 15;
}



__host__ void CUDA_simple_struct_copy_test(){

    //Wwwwhat the fff
    //hipMemcpy(device->storage,...)     does not work.
    //hipMemcpy(host->storage,...)  does,
    //no matter the direction.
    //So that means the pointer device->storage must first be copied to a simple
    //float *,
    // then copied to host.
    // ??? okay
    //Wait, no, that makes perfect sense.
    //if you call device.storage from the host,
    //the host tries to find...
    //wait, but why does it work when copying to a float *?

    const int N = 10;

    test_struct * host_input = new test_struct;
    host_input->test_int[5] = 10;
    host_input->storage = new float[N];
    for(int i = 0; i < N; i++){ host_input->storage[i] = i;};

    // hipMemset(dataGPU, 0, 1000*sizeof(int));

    //copy the struct, plus values on the stack
    test_struct * device;
    gpu_error_check(hipMalloc((void**)&device, sizeof(test_struct)));
    gpu_error_check(hipMemcpy(device, host_input, sizeof(test_struct), hipMemcpyHostToDevice));

    float * device_storage;
    gpu_error_check(hipMalloc(&device_storage, N*sizeof(float)));
    //copy the data
    gpu_error_check(hipMemcpy(device_storage, host_input->storage, N*sizeof(float), hipMemcpyHostToDevice));
    //bind - copy the pointer itself
    gpu_error_check(hipMemcpy(&(device->storage), &device_storage, sizeof(device->storage), hipMemcpyHostToDevice));

    test_fill_simple<<<1, 1>>>(device);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    float * host_output_storage = host_input->storage;
    //then copy the struct itself
    gpu_error_check(hipMemcpy(host_input, device, sizeof(test_struct), hipMemcpyDeviceToHost));
    //but of course now the host pointers are overwritten with device pointers, so we need to fix that:
    host_input->storage = host_output_storage;

    //copy the data back
    float * device_output_storage;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_output_storage,&(device->storage), sizeof(device->storage), hipMemcpyDeviceToHost));
    //then the data itself
    gpu_error_check(hipMemcpy(host_input->storage, device_output_storage, N*sizeof(float), hipMemcpyDeviceToHost));


    pretty_print_array(host_input->storage, 0, N);
    pretty_print_array(host_input->test_int, 0, N);

    ASSERT_NEAR(host_input->storage[0],10,1e-3);
    ASSERT_NEAR(host_input->test_int[5],15,1e-3);
    //ASSERT_NEAR(host_input->test_int[0],0,1e-3); //zero test case!


    // gpu_error_check(hipFree(device->storage));
    // gpu_error_check(hipFree(device));

    delete [] host_input->storage;
    delete host_input;
}

void construct_device_struct(test_struct ** device_struct){
    //construct the struct
    gpu_error_check(hipMalloc(device_struct, sizeof(test_struct)));

    float * device_storage;
    gpu_error_check(hipMalloc(&device_storage, 10*sizeof(* device_storage)));
    gpu_error_check(hipMemset(device_storage,0,10*sizeof(* device_storage))); //must memset the whole array!
    //copy pointer to array into struct
    gpu_error_check(hipMemcpy(&((**device_struct).storage), &device_storage, sizeof((**device_struct).storage), hipMemcpyHostToDevice));
}

void copy_to_device_struct(test_struct ** device_struct, test_struct ** host_struct){
    float * device_storage; //get the pointer from the device
    gpu_error_check(hipMemcpy(&device_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), hipMemcpyDeviceToHost));
    //and now copy the data.
    gpu_error_check(hipMemcpy(device_storage, (**host_struct).storage,  10*sizeof(* device_storage), hipMemcpyHostToDevice));

    //copy struct itself, wiping all the pointers,
    gpu_error_check(hipMemcpy(*device_struct, *host_struct, sizeof(test_struct), hipMemcpyHostToDevice));

    //then re-copy the pointers.
    gpu_error_check(hipMemcpy(&((**device_struct).storage), &device_storage, sizeof((**device_struct).storage), hipMemcpyHostToDevice));
    //There's a PCIe latency issue here, since we're going * -> host, data -> device,
    //but whatever!
}


void copy_to_host_struct(test_struct ** device_struct, test_struct ** host_struct){

    float * device_output_storage;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_output_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), hipMemcpyDeviceToHost));
    //then the data itself
    gpu_error_check(hipMemcpy((**host_struct).storage, device_output_storage, 10*sizeof(* device_output_storage), hipMemcpyDeviceToHost));

    //save for after the wipe
    float * host_storage = (**host_struct).storage;

    //copy struct itself, wiping all the host pointers,
    gpu_error_check(hipMemcpy(*host_struct, *device_struct, sizeof(test_struct), hipMemcpyDeviceToHost));

    (**host_struct).storage = host_storage;
}

__host__ void CUDA_simple_struct_copy_test_2(){

    const int N = 10;

    test_struct origin_host_struct;
    test_struct * host_struct = &origin_host_struct;

    (*host_struct).test_int[5] = 10;
    (*host_struct).storage = new float[N];
    for(int i = 0; i < N; i++){ (*host_struct).storage[i] = i;};

    test_struct * device_struct;
    construct_device_struct(&device_struct); //s
    //double pointer required to preserve malloc edit

    copy_to_device_struct(&device_struct, &host_struct);
    copy_to_device_struct(&device_struct, &host_struct); //check if our pointers were preserved

    test_fill_simple<<<1, 1>>>(device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    copy_to_host_struct(&device_struct, &host_struct);
    copy_to_host_struct(&device_struct, &host_struct);

    pretty_print_array(host_struct->storage, 0, N);
    pretty_print_array(host_struct->test_int, 0, N);

    ASSERT_NEAR(host_struct->storage[0],10,1e-3);
    ASSERT_NEAR(host_struct->test_int[5],15,1e-3);

    delete [] (*host_struct).storage;
    //copy the struct, plus values on the stack


    //
    // //copy the data
    // gpu_error_check(hipMemcpy(device_storage, host_input->storage, N*sizeof(float), hipMemcpyHostToDevice));
    // //bind - copy the pointer itself
    // gpu_error_check(hipMemcpy(&(device->storage), &device_storage, sizeof(device->storage), hipMemcpyHostToDevice));
    //
    // test_fill_simple<<<1, 1>>>(device);
    // gpu_error_check( hipPeekAtLastError() );
    // gpu_error_check( hipDeviceSynchronize() );
    //
    // delete [] host_input->storage;
    //
    // //then copy the struct itself
    // gpu_error_check(hipMemcpy(host_input, device, sizeof(test_struct), hipMemcpyDeviceToHost));
    // //but of course now the pointers are overwritten with device pointers, so we need to fix that:
    // float * host_output_storage = new float[N];
    // host_input->storage = host_output_storage;
    //
    // //copy the data back
    // float * device_output_storage;
    // //copy the pointer to the data
    // gpu_error_check(hipMemcpy(&device_output_storage,&(device->storage), sizeof(device->storage), hipMemcpyDeviceToHost));
    // //then the data itself
    // gpu_error_check(hipMemcpy(host_input->storage, device_output_storage, N*sizeof(float), hipMemcpyDeviceToHost));
    //
    //
    // pretty_print_array(host_input->storage, 0, N);
    // pretty_print_array(host_input->test_int, 0, N);
    //
    // ASSERT_NEAR(host_input->storage[0],10,1e-3);
    // ASSERT_NEAR(host_input->test_int[5],15,1e-3);
}



__host__ void CUDA_struct_copy_test(){
//     //cuda Unified Memory with ManagedMemcpy would be much simpler,
//     //but I think I prefer to see precisely when a PCIe transfer will occur.
//
//     int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
//     //input, which we don't want to mess with
//     physics_mesh * host_input = new physics_mesh(mesh_sizes, 1);
//     //so we create a surrogate to copy values to, without destructors
//     physics_mesh * host_b = new physics_mesh(1);
//     physics_mesh * device_a;
//
//     host_input->potential[10] = 100;
//     //copy all the physics arrays
//     POTENTIAL_TYPE * device_potential;
//     gpu_error_check(hipMalloc(&device_potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE)));
//                         //order is (device, host
//     gpu_error_check(hipMemcpy(device_potential, host_input->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyHostToDevice));
//
//     //this step has to be done before the struct is copied to the device
//     host_b->potential = device_potential;
//
//     //copy the struct itself (and accessory array values on the stack)
//     gpu_error_check(hipMalloc(&device_a, sizeof(physics_mesh)));
//     gpu_error_check(hipMemcpy(device_a, host_b, sizeof(physics_mesh), hipMemcpyHostToDevice));
//
//     test_fill<<<1, 1>>>(*device_a);
//     gpu_error_check( hipPeekAtLastError() );
//     gpu_error_check( hipDeviceSynchronize() );
//
//
//     // //and back again
//     // physics_mesh * host_output = new physics_mesh(1); // no destructors!
//     // gpu_error_check(hipMemcpy(host_output, device_a, sizeof(physics_mesh), hipMemcpyDeviceToHost));
//     POTENTIAL_TYPE * output_potential = new POTENTIAL_TYPE[MESH_BUFFER_SIZE];
//
//     gpu_error_check(hipMemcpy(output_potential, device_a->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyDeviceToHost));
//
//
//     hipFree(device_potential);
//     hipFree(device_a->potential);
//     hipFree(device_a);
//     delete host_b;
//     delete host_input;
//
//     // delete [] host_output->potential;
//     // delete host_output;
//     // delete [] device_a->potential;
//     delete [] output_potential;
//
}
//
