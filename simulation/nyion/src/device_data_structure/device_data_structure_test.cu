#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"
#include "host_data_structure.hpp"

//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!

__global__ void test_fill(physics_mesh &d_a) {
    for(int i = 0; i < 10; i++){
        d_a.potential[i] = i;
    }
}

__host__ void CUDA_simple_copy_test(){
    const int N = 10;

    float * x = new float[N];
    for(int i = 0; i < N; i++){ x[i]= i;};
    float * d_x;

    gpu_error_check(hipMalloc(&d_x, N*sizeof(float)));
                    //always dest, src
    gpu_error_check(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
    gpu_error_check(hipDeviceSynchronize());

    gpu_error_check(hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost));
    gpu_error_check( hipDeviceSynchronize() );

    pretty_print_array(x, 0, N);

    gpu_error_check(hipFree(d_x));

    delete [] x;
}

struct test_struct{
    int test_int[10];
    float * storage;
};

__global__ void test_fill_simple(test_struct * d_a) {
    for(int i = 0; i < 10; i++){
        d_a->storage[i] = d_a->test_int[5];
    }
    d_a->test_int[5] = 15;
}



__host__ void CUDA_simple_struct_copy_test(){



    // gpu_error_check(hipMalloc((void**)&(device->storage), N*sizeof(float)));
    // hipMemcpy(&(device->storage), &(d_arr), sizeof(int*), hipMemcpyHostToDevice); //copy the pointer value itself
    //
    const int N = 10;

    test_struct * host_input = new test_struct;
    host_input->test_int[5] = 10;
    host_input->storage = new float[N];
    for(int i = 0; i < N; i++){ host_input->storage[i] = i;};


    //copy the struct, plus values on the stack
    test_struct * device;
    gpu_error_check(hipMalloc((void**)&device, sizeof(test_struct)));
    gpu_error_check(hipMemcpy(device, host_input, sizeof(test_struct), hipMemcpyHostToDevice));

    float * device_storage;
    gpu_error_check(hipMalloc(&device_storage, N*sizeof(float)));
    //copy the data
    gpu_error_check(hipMemcpy(device_storage, host_input->storage, N*sizeof(float), hipMemcpyHostToDevice));
    //bind - copy the pointer itself
    gpu_error_check(hipMemcpy(&(device->storage), &device_storage, sizeof(device->storage), hipMemcpyHostToDevice));


    test_fill_simple<<<1, 1>>>(device);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    delete [] host_input->storage;

    //then copy the struct itself
    gpu_error_check(hipMemcpy(host_input, device, sizeof(test_struct), hipMemcpyDeviceToHost));
    //but of course now the pointers are overwritten with device pointers, so we need to fix that:
    float * host_output_storage = new float[N];
    host_input->storage = host_output_storage;

    //copy the data back
    float * device_output_storage;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_output_storage,&(device->storage), sizeof(device->storage), hipMemcpyDeviceToHost));
    //then the data itself
    gpu_error_check(hipMemcpy(host_input->storage, device_output_storage, N*sizeof(float), hipMemcpyDeviceToHost));



    pretty_print_array(host_input->storage, 0, N);
    pretty_print_array(host_input->test_int, 0, N);

    // //
    // gpu_error_check(hipMemcpy(host_input, device, sizeof(test_struct), hipMemcpyDeviceToHost));
    // gpu_error_check(hipMemcpy(host_input->storage, device->storage, N*sizeof(float), hipMemcpyDeviceToHost));
    //
    //
    //
    // gpu_error_check(hipFree(device->storage));
    // gpu_error_check(hipFree(device));
    //
    delete [] host_input->storage;
    delete host_input;
}


__host__ void CUDA_struct_copy_test(){
//     //cuda Unified Memory with ManagedMemcpy would be much simpler,
//     //but I think I prefer to see precisely when a PCIe transfer will occur.
//
//     int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
//     //input, which we don't want to mess with
//     physics_mesh * host_input = new physics_mesh(mesh_sizes, 1);
//     //so we create a surrogate to copy values to, without destructors
//     physics_mesh * host_b = new physics_mesh(1);
//     physics_mesh * device_a;
//
//     host_input->potential[10] = 100;
//     //copy all the physics arrays
//     POTENTIAL_TYPE * device_potential;
//     gpu_error_check(hipMalloc(&device_potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE)));
//                         //order is (device, host
//     gpu_error_check(hipMemcpy(device_potential, host_input->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyHostToDevice));
//
//     //this step has to be done before the struct is copied to the device
//     host_b->potential = device_potential;
//
//     //copy the struct itself (and accessory array values on the stack)
//     gpu_error_check(hipMalloc(&device_a, sizeof(physics_mesh)));
//     gpu_error_check(hipMemcpy(device_a, host_b, sizeof(physics_mesh), hipMemcpyHostToDevice));
//
//     test_fill<<<1, 1>>>(*device_a);
//     gpu_error_check( hipPeekAtLastError() );
//     gpu_error_check( hipDeviceSynchronize() );
//
//
//     // //and back again
//     // physics_mesh * host_output = new physics_mesh(1); // no destructors!
//     // gpu_error_check(hipMemcpy(host_output, device_a, sizeof(physics_mesh), hipMemcpyDeviceToHost));
//     POTENTIAL_TYPE * output_potential = new POTENTIAL_TYPE[MESH_BUFFER_SIZE];
//
//     gpu_error_check(hipMemcpy(output_potential, device_a->potential, MESH_BUFFER_SIZE*sizeof(POTENTIAL_TYPE), hipMemcpyDeviceToHost));
//
//
//     hipFree(device_potential);
//     hipFree(device_a->potential);
//     hipFree(device_a);
//     delete host_b;
//     delete host_input;
//
//     // delete [] host_output->potential;
//     // delete host_output;
//     // delete [] device_a->potential;
//     delete [] output_potential;
//
}
//
