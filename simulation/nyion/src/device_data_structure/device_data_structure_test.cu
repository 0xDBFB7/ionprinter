#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"
#include "host_data_structure.hpp"


#include "gtest/gtest.h"

//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck --leak-check full ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!


// cuda Unified Memory with ManagedMemcpy would be much simpler,
// but I think I prefer to see precisely when a PCIe transfer will occur.

//unified memory allows multiple GPUs, which might be useful down the road.


__global__ void physics_test_fill_simple(physics_mesh * d_a) {
    for(int i = 0; i < 10; i++){
        d_a->potential[i] = 100;
    }
}

__host__ void CUDA_physics_mesh_copy_test(){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;

    for(int i = 0; i < 10; i++){ origin_host.potential[i] = i;};

    physics_mesh * device_struct;

    physics_mesh::device_constructor(&device_struct); //s
    //double pointer required to preserve malloc edit

    physics_mesh::copy_to_device_struct(&device_struct, &host_struct);
    physics_mesh::copy_to_device_struct(&device_struct, &host_struct); //check if our pointers were preserved


    //run kernel
    physics_test_fill_simple<<<1, 1>>>(device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );


    physics_mesh::copy_to_host_struct(&device_struct, &host_struct);
    physics_mesh::copy_to_host_struct(&device_struct, &host_struct);

    pretty_print_array(origin_host.potential, 0, 20);

    origin_host.pretty_print();

    ASSERT_NEAR(origin_host.potential[0],100,1e-3);
    ASSERT_NEAR(origin_host.mesh_sizes[0],3,1e-3);
    //remember to test the zero case, to catch garbage on initialization

    physics_mesh::device_destructor(&device_struct);

}
