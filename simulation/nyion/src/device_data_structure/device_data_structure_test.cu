#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"
#include "host_data_structure.hpp"

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know!
__global__ void test_fill(physics_mesh * d_a) {
    
}



__host__ void CUDA_struct_copy_test(){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh a(mesh_sizes, 1);

    physics_mesh * d_a;

    gpu_error_check(hipMalloc(&d_a, sizeof(physics_mesh)));
    gpu_error_check(hipMemcpy(d_a, &a, sizeof(physics_mesh), hipMemcpyHostToDevice));
    gpu_error_check(hipMalloc(d_a->potential, sizeof(POTENTIAL_BUFFER_TYPE)));
    test_fill<<<1, 1>>>(&a);
    //<<<block # in level, cube(mesh_sizes[level])>>>
    //both can be dim3
    //cuda Unified Memory with ManagedMemcpy would be simpler,
    //but I think I prefer to see precisely when a PCIe transfer will occur.

    hipFree(d_a);
}
