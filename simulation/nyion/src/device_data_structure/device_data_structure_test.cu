#include "device_data_structure.hpp"
#include "host_data_structure.hpp"




void CUDA_struct_copy_test(){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh a(mesh_sizes, 1);

    physics_mesh * d_a;

    gpu_error_check(hipMalloc(&d_a, sizeof(physics_mesh)));
    hipMemcpy(d_a, &a, sizeof(physics_mesh), hipMemcpyHostToDevice);

    add_<<<1, 1>>>();
    //<<<block # in level, cube(mesh_sizes[level])>>>
    //both can be dim3

    hipFree(d_a);
}
