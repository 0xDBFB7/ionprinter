#include "hip/hip_runtime.h"
__global__ void test_fill(physics_mesh &d_a) {
    for(int i = 0; i < 10; i++){
        d_a.potential[i] = i;
    }
}

__host__ void CUDA_simple_copy_test(){
    const int N = 10;

    float * x = new float[N];
    for(int i = 0; i < N; i++){ x[i]= i;};
    float * d_x;

    gpu_error_check(hipMalloc(&d_x, N*sizeof(float)));
                    //always dest, src
    gpu_error_check(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
    gpu_error_check(hipDeviceSynchronize());

    gpu_error_check(hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost));
    gpu_error_check( hipDeviceSynchronize() );

    pretty_print_array(x, 0, N);

    gpu_error_check(hipFree(d_x));

    delete [] x;
}

struct test_struct{
    int test_int[10];
    float * storage;
    uint8_t * small_storage;
    //must try different size types
};

__global__ void test_fill_simple(test_struct * d_a) {
    for(int i = 0; i < 10; i++){
        d_a->storage[i] = d_a->test_int[5];
    }
    d_a->test_int[5] = 15;
}

//Wwwwhat the fff
//hipMemcpy(device->storage,...)     does not work.
//hipMemcpy(host->storage,...)  does,
//no matter the direction.
//So that means the pointer device->storage must first be copied to a simple
//float *,
// then copied to host.
// ??? okay
//Wait, no, that makes perfect sense.
//if you call device.storage from the host,
//the host tries to find...
//wait, but why does it work when copying to a float *?


__host__ void CUDA_simple_struct_copy_test(){

    const int N = 10;

    test_struct * host_input = new test_struct;
    host_input->test_int[5] = 10;
    host_input->storage = new float[N];
    for(int i = 0; i < N; i++){ host_input->storage[i] = i;};

    // hipMemset(dataGPU, 0, 1000*sizeof(int));

    //copy the struct, plus values on the stack
    test_struct * device;
    gpu_error_check(hipMalloc((void**)&device, sizeof(test_struct)));
    gpu_error_check(hipMemcpy(device, host_input, sizeof(test_struct), hipMemcpyHostToDevice));

    float * device_storage;
    gpu_error_check(hipMalloc(&device_storage, N*sizeof(float)));
    //copy the data
    gpu_error_check(hipMemcpy(device_storage, host_input->storage, N*sizeof(float), hipMemcpyHostToDevice));
    //bind - copy the pointer itself
    gpu_error_check(hipMemcpy(&(device->storage), &device_storage, sizeof(device->storage), hipMemcpyHostToDevice));

    test_fill_simple<<<1, 1>>>(device);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    float * host_output_storage = host_input->storage;
    //then copy the struct itself
    gpu_error_check(hipMemcpy(host_input, device, sizeof(test_struct), hipMemcpyDeviceToHost));
    //but of course now the host pointers are overwritten with device pointers, so we need to fix that:
    host_input->storage = host_output_storage;

    //copy the data back
    float * device_output_storage;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_output_storage,&(device->storage), sizeof(device->storage), hipMemcpyDeviceToHost));
    //then the data itself
    gpu_error_check(hipMemcpy(host_input->storage, device_output_storage, N*sizeof(float), hipMemcpyDeviceToHost));


    pretty_print_array(host_input->storage, 0, N);
    pretty_print_array(host_input->test_int, 0, N);

    ASSERT_NEAR(host_input->storage[0],10,1e-3);
    ASSERT_NEAR(host_input->test_int[5],15,1e-3);
    //ASSERT_NEAR(host_input->test_int[0],0,1e-3); //zero test case!


    // gpu_error_check(hipFree(device->storage));
    // gpu_error_check(hipFree(device));

    delete [] host_input->storage;
    delete host_input;
}

void construct_device_struct(test_struct ** device_struct){
    //construct the struct
    gpu_error_check(hipMalloc(device_struct, sizeof(test_struct)));

    float * device_storage;
    gpu_error_check(hipMalloc(&device_storage, 10*sizeof(* device_storage)));
    gpu_error_check(hipMemset(device_storage,0,10*sizeof(* device_storage))); //must memset the whole array!
    //copy pointer to array into struct
    gpu_error_check(hipMemcpy(&((**device_struct).storage), &device_storage, sizeof((**device_struct).storage), hipMemcpyHostToDevice));
}

void copy_to_device_struct(test_struct ** device_struct, test_struct ** host_struct){
    float * device_storage; //get the pointer from the device
    gpu_error_check(hipMemcpy(&device_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), hipMemcpyDeviceToHost));
    //and now copy the data.
    gpu_error_check(hipMemcpy(device_storage, (**host_struct).storage,  10*sizeof(* device_storage), hipMemcpyHostToDevice));

    //copy struct itself, wiping all the pointers,
    gpu_error_check(hipMemcpy(*device_struct, *host_struct, sizeof(test_struct), hipMemcpyHostToDevice));

    //if all the pointers were put to the end of physics_mesh, perhaps this wipe wouldn't occur?⁠
    //sizeof = &(struct) -  &(struct->canary)

    //then re-copy the pointers.
    gpu_error_check(hipMemcpy(&((**device_struct).storage), &device_storage, sizeof((**device_struct).storage), hipMemcpyHostToDevice));
    //There's a PCIe latency issue here, since we're going * -> host, data -> device,
    //but whatever!
}

void copy_to_host_struct(test_struct ** device_struct, test_struct ** host_struct){
    float * device_temp;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_temp, &((**device_struct).storage), sizeof(((**device_struct).storage)), hipMemcpyDeviceToHost));
    //then the data itself
    gpu_error_check(hipMemcpy((**host_struct).storage, device_temp, 10*sizeof(* device_temp), hipMemcpyDeviceToHost));

    //save for after the wipe
    float * host_temp = (**host_struct).storage;

    //copy struct itself, wiping all the host pointers,
    gpu_error_check(hipMemcpy(*host_struct, *device_struct, sizeof(test_struct), hipMemcpyDeviceToHost));

    //must be done simultaneously!
    (**host_struct).storage = host_temp;
}

void destruct_device_struct(test_struct ** device_struct){
    float * device_output_storage;
    //copy the pointer to the data
    gpu_error_check(hipMemcpy(&device_output_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), hipMemcpyDeviceToHost));
    //then destroy it!
    gpu_error_check(hipFree(device_output_storage));


    gpu_error_check(hipFree(&(**device_struct)));
}

__host__ void CUDA_simple_struct_copy_test_2(){

    const int N = 10;

    test_struct origin_host_struct;
    test_struct * host_struct = &origin_host_struct;

    (*host_struct).test_int[5] = 10;
    (*host_struct).storage = new float[N];
    for(int i = 0; i < N; i++){ (*host_struct).storage[i] = i;};

    test_struct * device_struct;
    construct_device_struct(&device_struct); //s
    //double pointer required to preserve malloc edit

    copy_to_device_struct(&device_struct, &host_struct);
    copy_to_device_struct(&device_struct, &host_struct); //check if our pointers were preserved

    test_fill_simple<<<1, 1>>>(device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    copy_to_host_struct(&device_struct, &host_struct);
    copy_to_host_struct(&device_struct, &host_struct);

    // pretty_print_array(host_struct->storage, 0, N);
    // pretty_print_array(host_struct->test_int, 0, N);

    ASSERT_NEAR(host_struct->storage[0],10,1e-3);
    ASSERT_NEAR(host_struct->test_int[5],15,1e-3);
    //remember to test the zero case, to catch garbage on initialization

    delete [] (*host_struct).storage;
    destruct_device_struct(&device_struct);
    //copy the struct, plus values on the stack
}
