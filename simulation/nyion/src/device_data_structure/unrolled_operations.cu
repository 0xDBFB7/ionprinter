#include "hip/hip_runtime.h"
/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

//
//Ryzen 7: 0.25 TF/40 gbps.
https://en.wikichip.org/wiki/amd/ryzen_7/1700
//
//GTX 1060 is 3 TFlops/160 gbps,
//An RTX 2070 Super hits 8.2 TF/448.0 gbps, and 21 to 51 TFlops half precision.
//V100 hits 14TF/900 +100TF half.
//https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
//
//
*/

#include "unrolled_operations.hpp"
#include "host_data_structure.hpp"

//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!


#define N 10

#define MAX 100

struct test_struct{
    int test_int = 0;
    __device__ void increment(){
        test_int++;
    }
};

// __device__ float hiprand_uniform (hiprandState_t *state)
// __device__ float hiprand_normal (hiprandState_t *state)
//                  hiprand(&state)
__global__ void add(test_struct * d_a) {
    d_a->increment();
  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
}



void test_cuda_struct(){
    test_struct * a;
    test_struct * d_a;

    gpu_error_check( hipMalloc(&d_a, sizeof(test_struct)));
    hipMemcpy(d_a, a, sizeof(test_struct), hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a);

    hipFree(d_a);
    free(a);
}


void test_cuda(float * x)
{

  // const int N = 10;
  // float *d_x, *d_y;
  // //
  // gpu_error_check( hipMalloc(&d_x, N*sizeof(float)));
  // gpu_error_check( hipMalloc(&d_y, N*sizeof(float)) );
  // //
  // // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  // // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  test_struct * a;
  test_struct * d_a;

  gpu_error_check( hipMalloc(&a, sizeof(test_struct)));
  hipMemcpy(d_a, a, sizeof(test_struct), hipMemcpyHostToDevice);
  // Perform SAXPY on 1M elements
  // gpu_error_check( add<<<1, 1>>>(); );

  // gpu_error_check( add<<<1, 1>>>(); );

  // DisplayHeader();
  // hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  //
  // hipFree(d_x);
  // hipFree(d_y);
}
