#include "hip/hip_runtime.h"
/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

//
//Ryzen 7: 0.25 TF/40 gbps.
https://en.wikichip.org/wiki/amd/ryzen_7/1700
//
//GTX 1060 is 3 TFlops/160 gbps,
//An RTX 2070 Super hits 8.2 TF/448.0 gbps, and 21 to 51 TFlops half precision.
//V100 hits 14TF/900 +100TF half.
//https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
//
//
*/

#include "unrolled_operations.hpp"

//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!


#define N 10

#define MAX 100


// __device__ float hiprand_uniform (hiprandState_t *state)
// __device__ float hiprand_normal (hiprandState_t *state)
//                  hiprand(&state)
__global__
void add()
{
  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
}


void test_cuda(float * x)
{

  // const int N = 10;
  // float *d_x, *d_y;
  // //
  // gpu_error_check( hipMalloc(&d_x, N*sizeof(float)));
  // gpu_error_check( hipMalloc(&d_y, N*sizeof(float)) );
  // //
  // // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  // // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // gpu_error_check( add<<<1, 1>>>(); );


  // DisplayHeader();
  // hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  //
  // hipFree(d_x);
  // hipFree(d_y);
}
