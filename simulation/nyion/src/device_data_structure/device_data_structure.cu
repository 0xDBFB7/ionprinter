
#include <hip/hip_runtime.h>
// /*
//
// Having a tree relationship between with levels and blocks makes some kind of sense.
// Everything multigrid is naturally tree-based,
// having a structure to traverse is great for cell-world and world-cell lookups,
// and to generate the linkages between ghosts.
// It's also easier to code the construction of meshes.
//
// However, traversing the tree takes jumps and is generally poorly optimized,
// especially on a GPU, where a jump stalls the warp.
// Once the trees are established on the heap, however, a simple list of indices
// can be constructed to traverse linearly.
//
// It might be possible to do away with the tree entirely, but
// trying to visualize 'neighbors' with arbitrarily placed blocks
// does my head in.
//
// It's important to note that this entire implementation completely disregards
// the cache altogether. The indice system means the next block could be halfway to Manitoba
// out of cache; there's no cache concurrency, congruency, or congealency; the
// &mesh argument means we're probably bringing 20x as much data along as we need for each function;
// etc.
//
// If one were truly smart, one could probably eek the same performance out of a
// homogenous computing system. I am not.
//
// //block_list
// //block_sizes
// //connections
//
// //
// //Ryzen 7: 0.25 TF/40 gbps. https://en.wikichip.org/wiki/amd/ryzen_7/1700
// //GTX 1060: 3 TFlops/160 GBps.
// //RTX 2070 Super: 8.2 TF/448.0 GBps - 21 to 51 TFlops half precision.
// //V100 hits 14TF/900 GBps + 100TF half.
// //https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
// //
// //Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!
// //
// */
//
// #include "device_data_structure.hpp"
// #include "host_data_structure.hpp"
//
// //
// //
// // #define CONSTRUCTOR_MACRO(TYPE, NAME, SIZE)     \
// //     TYPE * device_temp;         \
// //     gpu_error_check(cudaMalloc(&device_storage, (SIZE)*sizeof(* device_temp)));\
// //     gpu_error_check(cudaMemset(device_storage,0,(SIZE)*sizeof(* device_temp))); \//must memset the whole array!
// //     ;\ //copy pointer to array into struct
// //     gpu_error_check(cudaMemcpy(&((**device_struct).NAME), &device_temp, sizeof((**device_struct).NAME), cudaMemcpyHostToDevice));
// //
//
// void physics_mesh::device_constructor(physics_mesh ** device_struct){
//     //construct the struct itself
//     gpu_error_check(cudaMalloc(device_struct, sizeof(physics_mesh)));
//
// }
//
// void physics_mesh::copy_to_device_struct(test_struct ** device_struct, test_struct ** host_struct){
//     float * device_storage; //get the pointer from the device
//     gpu_error_check(cudaMemcpy(&device_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), cudaMemcpyDeviceToHost));
//     //and now copy the data.
//     gpu_error_check(cudaMemcpy(device_storage, (**host_struct).storage,  10*sizeof(* device_storage), cudaMemcpyHostToDevice));
//
//     //copy struct itself, wiping all the pointers,
//     gpu_error_check(cudaMemcpy(*device_struct, *host_struct, sizeof(test_struct), cudaMemcpyHostToDevice));
//
//     //then re-copy the pointers.
//     gpu_error_check(cudaMemcpy(&((**device_struct).storage), &device_storage, sizeof((**device_struct).storage), cudaMemcpyHostToDevice));
//     //There's a PCIe latency issue here, since we're going * -> host, data -> device,
//     //but whatever!
// }
//
// void physics_mesh::copy_to_host_struct(test_struct ** device_struct, test_struct ** host_struct){
//     float * device_temp;
//     //copy the pointer to the data
//     gpu_error_check(cudaMemcpy(&device_temp, &((**device_struct).storage), sizeof(((**device_struct).storage)), cudaMemcpyDeviceToHost));
//     //then the data itself
//     gpu_error_check(cudaMemcpy((**host_struct).storage, device_temp, 10*sizeof(* device_temp), cudaMemcpyDeviceToHost));
//
//     //save for after the wipe
//     float * host_temp = (**host_struct).storage;
//
//     //copy struct itself, wiping all the host pointers,
//     gpu_error_check(cudaMemcpy(*host_struct, *device_struct, sizeof(test_struct), cudaMemcpyDeviceToHost));
//
//     //must be done simultaneously!
//     (**host_struct).storage = host_storage;
// }
//
// void physics_mesh::destruct_device_struct(test_struct ** device_struct){
//     float * device_output_storage;
//     //copy the pointer to the data
//     gpu_error_check(cudaMemcpy(&device_output_storage, &((**device_struct).storage), sizeof(((**device_struct).storage)), cudaMemcpyDeviceToHost));
//     //then destroy it!
//     gpu_error_check(cudaFree(device_output_storage));
//
//
//     gpu_error_check(cudaFree(&(**device_struct)));
// }
// //
//
//
//
//
// void test_cuda(float * x){
// }
