/*

Having a tree relationship between with levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups,
and to generate the linkages between ghosts.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized,
especially on a GPU, where a jump stalls the warp.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.

It might be possible to do away with the tree entirely, but
trying to visualize 'neighbors' with arbitrarily placed blocks
does my head in.

It's important to note that this entire implementation completely disregards
the cache altogether. The indice system means the next block could be halfway to Manitoba
out of cache; there's no cache concurrency, congruency, or congealency; the
&mesh argument means we're probably bringing 20x as much data along as we need for each function;
etc.

If one were truly smart, one could probably eek the same performance out of a
homogenous computing system. I am not.

//block_list
//block_sizes
//connections

//
//Ryzen 7: 0.25 TF/40 gbps. https://en.wikichip.org/wiki/amd/ryzen_7/1700
//GTX 1060: 3 TFlops/160 GBps.
//RTX 2070 Super: 8.2 TF/448.0 GBps - 21 to 51 TFlops half precision.
//V100 hits 14TF/900 GBps + 100TF half.
//https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
//
//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!
//
*/

//if all the pointers were put to the end of physics_mesh, perhaps we could simplify the pointer wipe code?
//hipMemcpy(...,&(struct) -  &(struct->canary)...)
//
// oh, no, structure order doesn't seem to be preserved in the case of

#include "device_data_structure.hpp"
#include "host_data_structure.hpp"


#define CONSTRUCTOR_MACRO(TYPE, NAME, N)     \
    TYPE * device_temp;         \
    /* malloc an array on the device, */ \
    gpu_error_check(hipMalloc(&device_temp, (N)*sizeof(* device_temp))); \
    /* memset the whole array, not just a buffer_end_pointer's worth */ \
    gpu_error_check(hipMemset(device_temp,0,(N)*sizeof(* device_temp))); \
    /* then copy the pointer to the new array into the struct. */ \
    gpu_error_check(hipMemcpy(&((**device_struct).NAME), &device_temp, sizeof((**device_struct).NAME), hipMemcpyHostToDevice));


void physics_mesh::device_constructor(physics_mesh ** device_struct){
    //construct the struct itself
    gpu_error_check(hipMalloc(device_struct, sizeof(physics_mesh)));

    //construct the arrays and copy to the GPU
    CONSTRUCTOR_MACRO(float, potential, MESH_BUFFER_SIZE);
}




#define COPY_ARRAY_TO_DEVICE_MACRO(TYPE, NAME, N)     \
    TYPE * NAME; \
    /* get the pointer from the device */ \
    gpu_error_check(hipMemcpy(&NAME, &((**device_struct).NAME), sizeof(((**device_struct).NAME)), hipMemcpyDeviceToHost)); \
    /* and now copy the data. */ \
    gpu_error_check(hipMemcpy(NAME, (**host_struct).NAME,  (N)*sizeof(* NAME), hipMemcpyHostToDevice));


#define COPY_POINTERS_BACK_TO_DEVICE(NAME)     \
    gpu_error_check(hipMemcpy(&((**device_struct).NAME), &NAME, sizeof((**device_struct).NAME), hipMemcpyHostToDevice));


void physics_mesh::copy_to_device(physics_mesh ** device_struct, physics_mesh ** host_struct){
    //double pointer required to preserve malloc edit
    uint32_t length = (**host_struct).buffer_end_pointer;

    //copy array and also save a copy of each array pointer for later
    COPY_ARRAY_TO_DEVICE_MACRO(POTENTIAL_TYPE, potential, length);

    //copy struct itself, wiping all the pointers,
    gpu_error_check(hipMemcpy(*device_struct, *host_struct, sizeof(physics_mesh), hipMemcpyHostToDevice));

    //then re-copy the pointers.
    COPY_POINTERS_BACK_TO_DEVICE(potential);

}
//There's a PCIe latency thing here, since we're going * -> host, data -> device,
//but whatever!




#define COPY_ARRAY_TO_HOST_MACRO(TYPE, NAME, N) \
    TYPE * NAME_temp;    \
    /* copy the pointer to the data */  \
    gpu_error_check(hipMemcpy(&NAME_temp, &((**device_struct).NAME), sizeof(((**device_struct).NAME)), hipMemcpyDeviceToHost)); \
    /*  then the data itself */ \
    gpu_error_check(hipMemcpy((**host_struct).NAME, NAME_temp, N*sizeof(* NAME_temp), hipMemcpyDeviceToHost));



void physics_mesh::copy_to_host(physics_mesh ** device_struct, physics_mesh ** host_struct){

    //save host array pointers for after the wipe
    float * host_temp = (**host_struct).potential;

    //copy struct itself, wiping all the host pointers to arrays,
    gpu_error_check(hipMemcpy(*host_struct, *device_struct, sizeof(physics_mesh), hipMemcpyDeviceToHost));

    //then restore host pointers to host arrays
    (**host_struct).potential = host_temp;

    COPY_ARRAY_TO_HOST_MACRO(POTENTIAL_TYPE, potential, ((**host_struct).buffer_end_pointer));

}



#define DEFENESTRATE_ARRAY(TYPE, NAME) \
    TYPE * NAME; \
    /* //copy the pointer to the data */ \
    gpu_error_check(hipMemcpy(&NAME, &((**device_struct).NAME), sizeof(((**device_struct).NAME)), hipMemcpyDeviceToHost)); \
    /* //then destroy it! */ \
    gpu_error_check(hipFree(NAME)); \

void physics_mesh::device_destructor(physics_mesh ** device_struct){

    DEFENESTRATE_ARRAY(POTENTIAL_TYPE, potential);

    //then destroy the struct itself.
    gpu_error_check(hipFree(&(**device_struct)));
}



void test_cuda(float * x){
}
