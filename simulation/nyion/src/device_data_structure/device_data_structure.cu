#include "hip/hip_runtime.h"
/*

Having a tree relationship between with levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups,
and to generate the linkages between ghosts.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized,
especially on a GPU, where a jump stalls the warp.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.

It might be possible to do away with the tree entirely, but
trying to visualize 'neighbors' with arbitrarily placed blocks
does my head in.

It's important to note that this entire implementation completely disregards
the cache altogether. The indice system means the next block could be halfway to Manitoba
out of cache; there's no cache concurrency, congruency, or congealency; the
&mesh argument means we're probably bringing 20x as much data along as we need for each function;
etc.

If one were truly smart, one could probably eek the same performance out of a
homogenous computing system. I am not.

//block_list
//block_sizes
//connections

//
//Ryzen 7: 0.25 TF/40 gbps. https://en.wikichip.org/wiki/amd/ryzen_7/1700
//GTX 1060: 3 TFlops/160 GBps.
//RTX 2070 Super: 8.2 TF/448.0 GBps - 21 to 51 TFlops half precision.
//V100 hits 14TF/900 GBps + 100TF half.
//https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
//
//
*/

#include "device_data_structure.hpp"
#include "host_data_structure.hpp"

//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!


#define N 10

#define MAX 100

// __device__ float hiprand_uniform (hiprandState_t *state)
// __device__ float hiprand_normal (hiprandState_t *state)
//                  hiprand(&state)
__global__ void add_() {

  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
}

__host__ void construct_device_physics_mesh(physics_mesh * device_physics_mesh){
    gpu_error_check(hipMalloc((void**)&device_physics_mesh, sizeof(physics_mesh)));


}

// 
// __host__ void copy_physics_mesh_to_gpu(physics_mesh * host_physics_mesh, physics_mesh * device_physics_mesh){
//     //remember to zero allocated arrays!
//     //override new with zero constructor?
//
//     //copy struct and data on stack
//     gpu_error_check(hipMemcpy(device_physics_mesh, host_physics_mesh, sizeof(physics_mesh), hipMemcpyHostToDevice));
//
//
//     float * device_storage;
//     gpu_error_check(hipMalloc(&device_storage, N*sizeof(float)));
//     // //copy the data
//     gpu_error_check(hipMemcpy(device_storage, host_input->storage, N*sizeof(float), hipMemcpyHostToDevice));
//     // //bind - copy the pointer itself
//     gpu_error_check(hipMemcpy(&(device->storage), &device_storage, sizeof(device->storage), hipMemcpyHostToDevice));
// }
//
//


void test_cuda(float * x){
}
