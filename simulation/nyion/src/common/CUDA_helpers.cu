
#include "CUDA_helpers.hpp"

void display_GPU_info(){
    //from https://forge.univ-poitiers.fr/projects/calculateur-hulk/wiki/CUDA_toolkit
    //Thanks y'all!
    const int kb = 1024;
    const int mb = kb * kb;
    std::wcout << "" << std::endl << "=========" << std::endl << std::endl;

    std::wcout << "CUDA version:   v" << CUDART_VERSION << std::endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    std::wcout << "CUDA Devices: " << std::endl << std::endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
        std::wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
        std::wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
        std::wcout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

        std::wcout << "  Warp size:         " << props.warpSize << std::endl;
        std::wcout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << std::endl;
        std::wcout << std::endl;
    }
    std::wcout << "" << std::endl << "=========" << std::endl << std::endl;

}
