#include "hip/hip_runtime.h"
#include "unrolled_operations.hpp"

//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!


#define N 10

#define MAX 100


/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* numbers) {
  /* hiprand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand_uniform(&states[blockIdx.x]) * MAX;
}

int random_numbers() {
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<N, 1>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  unsigned int cpu_nums[N];
  unsigned int* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

  /* invoke the kernel to get some random numbers */
  randoms<<<N, 1>>>(states, gpu_nums);

  /* copy the random numbers back */
  hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

  /* print them out */
  for (int i = 0; i < N; i++) {
    printf("%u\n", cpu_nums[i]);
  }

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}
