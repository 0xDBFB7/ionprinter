//#include "unrolled_operations.hpp"
// #include "nyion.hpp"

/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

*/

// #include "unrolled_operations.hpp"


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define gpu_error_check(ans) { gpuAssert((ans), __FILE__, __LINE__); } //thanks to talonmies!
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPU error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void add()
{
  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
}

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    wcout << "CUDA version:   v" << CUDART_VERSION << endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}

void test_cuda()
{

  // const int N = 10;
  // float *d_x, *d_y;
  // //
  // gpu_error_check( cudaMalloc(&d_x, N*sizeof(float)));
  // gpu_error_check( cudaMalloc(&d_y, N*sizeof(float)) );
  // //
  // // cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
  // // cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // gpu_error_check( add<<<1, 1>>>(); );
  // printf("aaaaaaaaaaaaaaaaaaaa\n");

  DisplayHeader();
  // cudaMemcpy(x, d_x, N*sizeof(float), cudaMemcpyDeviceToHost);
  // cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);
  //
  // cudaFree(d_x);
  // cudaFree(d_y);
}
