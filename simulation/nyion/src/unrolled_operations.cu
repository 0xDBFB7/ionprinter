#include "hip/hip_runtime.h"
//#include "unrolled_operations.hpp"
#include "nyion.hpp"

/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

*/

#define gpu_error_check(ans) { gpuAssert((ans), __FILE__, __LINE__); } //thanks to talonmies!
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPU error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void add()
{
  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
  printf("test\n");
}


void test_cuda(float * x, float * y)
{

  // int N = 10;
  // float *d_x, *d_y;
  // //
  // gpu_error_check( hipMalloc(&d_x, N*sizeof(float)));
  // hipMalloc(&d_y, N*sizeof(float));
  // //
  // // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  // // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  add<<<1, 1>>>();


  // hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  //
  // hipFree(d_x);
  // hipFree(d_y);
}
