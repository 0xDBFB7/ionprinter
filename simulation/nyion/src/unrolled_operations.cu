#include "hip/hip_runtime.h"
#include "unrolled_operations.hpp"

/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

*/

__global__
void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int test_cuda(){
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();

  return 0;
}
