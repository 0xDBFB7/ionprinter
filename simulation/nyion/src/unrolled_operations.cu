#include "hip/hip_runtime.h"
/*

Here's the situation.

Having a tree relationship between levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups, etc.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.


//block_list
//block_sizes
//connections

*/

#include "unrolled_operations.hpp"

//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!


#define N 10

#define MAX 100

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* numbers) {
  /* hiprand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand_uniform(&states[blockIdx.x]) * MAX;
}

int random_numbers() {
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<N, 1>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  unsigned int cpu_nums[N];
  unsigned int* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

  /* invoke the kernel to get some random numbers */
  randoms<<<N, 1>>>(states, gpu_nums);

  /* copy the random numbers back */
  hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

  /* print them out */
  for (int i = 0; i < N; i++) {
    printf("%u\n", cpu_nums[i]);
  }

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}
//
// __device__ float hiprand_uniform (hiprandState_t *state)
// __device__ float hiprand_normal (hiprandState_t *state)
//                  hiprand(&state)
__global__
void add()
{
  // int i = blockIdx.x*blockDim.x + threadIdx.x;
  // x[0] = 0;
  // y[0] = 100;
}


void test_cuda(float * x)
{

  // const int N = 10;
  // float *d_x, *d_y;
  // //
  // gpu_error_check( hipMalloc(&d_x, N*sizeof(float)));
  // gpu_error_check( hipMalloc(&d_y, N*sizeof(float)) );
  // //
  // // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  // // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // gpu_error_check( add<<<1, 1>>>(); );

  random_numbers();

  // DisplayHeader();
  // hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  //
  // hipFree(d_x);
  // hipFree(d_y);
}
