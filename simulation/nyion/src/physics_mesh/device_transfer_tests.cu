#include "hip/hip_runtime.h"
#include "device_transfer_tests.hpp"

#include "nyion.hpp"

#include "physics_mesh.hpp"

__global__ void physics_test_fill_simple(physics_mesh &mesh) {
    for(int i = 0; i < mesh.buffer_end_pointer; i++){
        mesh.potential[i] += i+100;
        mesh.boundary_conditions[i] += i+100;

    }
}

TEST(CUDA, physics_mesh_device_copy_test){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;
//
    for(int i = 0; i < origin_host.buffer_end_pointer; i++){ origin_host.potential[i] = 10+i;};
    for(int i = 0; i < origin_host.buffer_end_pointer; i++){ origin_host.boundary_conditions[i] = 10+i;};

    physics_mesh * device_struct;

    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);
    physics_mesh::copy_to_device(&device_struct, &host_struct);
    //we do this twice to check if our pointers were preserved correctly - seperate into other test

    //run kernel
    physics_test_fill_simple<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);
    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    // pretty_print_array(origin_host.potential, 0, 20);
    // origin_host.pretty_print();

    ASSERT_NEAR(origin_host.potential[0],110,1e-3);
    ASSERT_NEAR(origin_host.boundary_conditions[0],110,1e-3);
    ASSERT_NEAR(origin_host.potential[origin_host.buffer_end_pointer+1],0,1e-3);
    //testing the zero case, to catch garbage on initialization
    ASSERT_NEAR(origin_host.mesh_sizes[0],3,1e-3);

    physics_mesh::device_destructor(&device_struct);
}




__global__ void refine_on_device(physics_mesh &mesh) {
    mesh.refine_cell(0,0);
}

TEST(CUDA, CUDA_refine_on_device){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;

    physics_mesh * device_struct;
    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);

    //run kernel
    refine_on_device<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    ASSERT_EQ(origin_host.refined_indices[0],cube(3));

    physics_mesh::device_destructor(&device_struct);
}


void link_cuda(); //forces CMAKE to link cuda test code.
                    // there's probably a much more elegant way to do this.
