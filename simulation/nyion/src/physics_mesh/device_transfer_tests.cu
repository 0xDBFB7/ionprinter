#include "hip/hip_runtime.h"
#include "nyion.hpp"

#include "physics_mesh.hpp"
// #include "traverse_state.hpp"
// #include "struct_transfer_helpers.hpp"



//make -j16 && /usr/local/cuda-10.2/bin/nvprof ./test/nyion_test
//make -j16 && /usr/local/cuda-10.2/bin/cuda-memcheck --leak-check full ./test/nyion_test

//https://stackoverflow.com/questions/9309195/copying-a-struct-containing-pointers-to-cuda-device
//the input to a kernel can actually be a host structure
//as long as all the pointers therein are to device arrays.
//good to know! see also
// --- Destructor (correct version)
//  __host__ __device__ ~Lock(void) {
//#if !defined(__HIPCC__)
//      gpuErrchk(hipFree(d_state));
//#else
//
//#endif
//  }
//note:
//Max block dimensions: [ 1024, 1024, 64 ]
//Max grid dimensions:  [ 2147483647, 65535, 65535 ]
// the grid is fine, but we might have more than 1024 blocks!


// cuda Unified Memory with ManagedMemcpy would be much simpler,
// but I think I prefer to see precisely when a PCIe transfer will occur.

//unified memory allows multiple GPUs, which might be useful down the road.


__global__ void physics_test_fill_simple(physics_mesh &mesh) {
    for(int i = 0; i < mesh.buffer_end_pointer; i++){
        mesh.potential[i] = i+100;
    }
}

void CUDA_physics_mesh_copy(){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;
//
    for(int i = 0; i < 10; i++){ origin_host.potential[i] = i;};

    physics_mesh * device_struct;

    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);
    physics_mesh::copy_to_device(&device_struct, &host_struct);
    //we do this twice to check if our pointers were preserved correctly - seperate into other test

    //run kernel
    physics_test_fill_simple<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);
    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    pretty_print_array(origin_host.potential, 0, 20);

    origin_host.pretty_print();

    ASSERT_NEAR(origin_host.potential[0],100,1e-3);
    ASSERT_NEAR(origin_host.potential[origin_host.buffer_end_pointer+1],0,1e-3);
    //testing the zero case, to catch garbage on initialization
    ASSERT_NEAR(origin_host.mesh_sizes[0],3,1e-3);

    physics_mesh::device_destructor(&device_struct);

}
