#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"

#include "device_transfer_tests.hpp"

#include "gtest/gtest.h"

#include "nyion.hpp"


__global__ void physics_test_fill_simple(physics_mesh &mesh) {
    for(int i = 0; i < mesh.buffer_end_pointer; i++){
        mesh.potential[i] += i+100;
        mesh.boundary_conditions[i] += i+100;

    }
}

TEST(CUDA, physics_mesh_device_copy_test){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;
//
    for(int i = 0; i < origin_host.buffer_end_pointer; i++){ origin_host.potential[i] = 10+i;};
    for(int i = 0; i < origin_host.buffer_end_pointer; i++){ origin_host.boundary_conditions[i] = 10+i;};

    physics_mesh * device_struct;

    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);
    physics_mesh::copy_to_device(&device_struct, &host_struct);
    //we do this twice to check if our pointers were preserved correctly - seperate into other test

    //run kernel
    physics_test_fill_simple<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);
    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    // pretty_print_array(origin_host.potential, 0, 20);
    // origin_host.pretty_print();

    ASSERT_NEAR(origin_host.potential[0],110,1e-3);
    ASSERT_NEAR(origin_host.boundary_conditions[0],110,1e-3);
    ASSERT_NEAR(origin_host.potential[origin_host.buffer_end_pointer+1],0,1e-3);
    //testing the zero case, to catch garbage on initialization
    ASSERT_NEAR(origin_host.mesh_sizes[0],3,1e-3);

    physics_mesh::device_destructor(&device_struct);
}

TEST(CUDA, physics_mesh_device_copy_test2){

    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;
//
    for(int i = 0; i < origin_host.buffer_end_pointer; i++){ origin_host.potential[i] = 10+i;};

    physics_mesh * device_struct;

    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);
    //we do this twice to check if our pointers were preserved correctly - seperate into other test

    //run kernel
    physics_test_fill_simple<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);
    physics_mesh::copy_to_host(&device_struct, &host_struct);


    ASSERT_NEAR(origin_host.potential[0],110,1e-3);

    physics_mesh::device_destructor(&device_struct);
}



__global__ void refine_on_device(physics_mesh &mesh) {
    mesh.refine_cell(0,0);
}

TEST(CUDA, CUDA_refine_on_device){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {3, 5, 5};
    physics_mesh origin_host(mesh_sizes, 1);
    physics_mesh * host_struct = &origin_host;

    physics_mesh * device_struct;
    physics_mesh::device_constructor(&device_struct);

    physics_mesh::copy_to_device(&device_struct, &host_struct);

    //run kernel
    refine_on_device<<<1, 1>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );

    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    ASSERT_EQ(origin_host.refined_indices[0],cube(3));

    physics_mesh::device_destructor(&device_struct);
}


TEST(CUDA, CUDA_device_copy_ghost_values){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {4,5};
    physics_mesh origin_host(mesh_sizes,2);

    origin_host.refine_cell(0, 21); //refine two non-ghost blocks
    origin_host.refine_cell(0, 22); //adjacent in +x
    origin_host.refine_cell(0, 37); //adjacent in +z
    origin_host.set_level_ghost_linkages(1);

    origin_host.potential[220] = 3.14;
    origin_host.potential[280] = 3.14*2;
    origin_host.potential[345] = 3.14*3;

    physics_mesh * host_struct = &origin_host;
    physics_mesh * device_struct;
    physics_mesh::device_constructor(&device_struct);
    physics_mesh::copy_to_device(&device_struct, &host_struct);

    physics_mesh::device_copy_ghost_values(host_struct, device_struct, &((*device_struct).potential), 1);

    physics_mesh::copy_to_host(&device_struct, &host_struct);

    hipDeviceSynchronize();

    ASSERT_NEAR(origin_host.potential[98],3.14,1e-2); //points to 220
    ASSERT_NEAR(origin_host.potential[158],3.14*2,1e-2); //points to 280
    ASSERT_NEAR(origin_host.potential[170],3.14*3,1e-2); //points to 345
    physics_mesh::device_destructor(&device_struct);
}

TEST(CUDA, CUDA_device_jacobi_kernel_1){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {4,5};
    physics_mesh origin_host(mesh_sizes,2);

    origin_host.potential[21] = 1;

    physics_mesh * host_struct = &origin_host;
    physics_mesh * device_struct;
    physics_mesh::device_constructor(&device_struct);
    physics_mesh::copy_to_device(&device_struct, &host_struct);

    physics_mesh::device_jacobi_relax(host_struct, device_struct, &((*device_struct).potential), 0);

    physics_mesh::copy_to_host(&device_struct, &host_struct);

    ASSERT_NEAR(origin_host.potential[22],(1.0/6.0),1e-3); //points to 220
}


TEST(CUDA, CUDA_size_blocks_1){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {4};
    physics_mesh origin_host(mesh_sizes,1);
    physics_mesh * host_struct = &origin_host;

    dim3 threads;
    dim3 blocks;
    set_GPU_dimensions(host_struct,blocks,threads,0);

    ASSERT_EQ(blocks.PHYSICAL_BLOCKS,1);
    ASSERT_EQ(blocks.SUB_BLOCKS,1);

    ASSERT_EQ(threads.x,2); //-2 ghosts
    ASSERT_EQ(threads.y,2);
    ASSERT_EQ(threads.z,2);
}


TEST(CUDA, CUDA_size_blocks_2){
    int mesh_sizes[MESH_BUFFER_DEPTH] = {34};
    physics_mesh origin_host(mesh_sizes,1);
    physics_mesh * host_struct = &origin_host;

    dim3 threads;
    dim3 blocks;
    set_GPU_dimensions(host_struct,blocks,threads,0);

    ASSERT_EQ(blocks.PHYSICAL_BLOCKS,1);
    ASSERT_EQ(blocks.SUB_BLOCKS,32);

    ASSERT_EQ(threads.x,32);
    ASSERT_EQ(threads.y,32);
    ASSERT_EQ(threads.z,1);
}


//
// TEST(CUDA, CUDA_device_jacobi_kernel_benchmark){
//     int mesh_sizes[MESH_BUFFER_DEPTH] = {50,5};
//     physics_mesh origin_host(mesh_sizes,2);
//
//     origin_host.potential[21] = 1;
//
//     physics_mesh * host_struct = &origin_host;
//     physics_mesh * device_struct;
//     physics_mesh::device_constructor(&device_struct);
//     physics_mesh::copy_to_device(&device_struct, &host_struct);
//
//     auto start = std::chrono::high_resolution_clock::now();
//
//     for(int i = 0; i < 10; i++){
//         physics_mesh::device_jacobi_relax(host_struct, device_struct, &((*device_struct).potential), 0);
//     }
//
//     auto end = std::chrono::high_resolution_clock::now();
//     auto duration = std::chrono::duration_cast<std::chrono::microseconds>( end-start ).count()/100.0;
//     std::cout << duration << " us, " << "\n";
//
//     physics_mesh::copy_to_host(&device_struct, &host_struct);
//
//     ASSERT_NEAR(origin_host.potential[22],1/6,1e-4); //points to 220
// }


void link_cuda(); //forces CMAKE to link cuda test code.
                    // there's probably a much more elegant way to do this.
