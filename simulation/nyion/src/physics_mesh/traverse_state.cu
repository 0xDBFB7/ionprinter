#include "hip/hip_runtime.h"
#include "traverse_state.hpp"

traverse_state::traverse_state(){
    current_depth = 0;
    block_beginning_indice = 0;
    current_indice = 0;
    started_traverse = true;

    for(int i = 0; i < MESH_BUFFER_DEPTH; i++){
        x_queue[i] = 0;
        y_queue[i] = 0;
        z_queue[i] = 0;
        ref_queue[i] = 0;
    }
}



bool traverse_state::equals(traverse_state &state_2, int depth){
    bool e_s = true;

    e_s = e_s && (current_depth == state_2.current_depth);
    e_s = e_s && (block_beginning_indice == state_2.block_beginning_indice);
    e_s = e_s && (current_indice == state_2.current_indice);
    e_s = e_s && (get_x() == state_2.get_x());
    e_s = e_s && (get_y() == state_2.get_y());
    e_s = e_s && (get_z() == state_2.get_z());

    for(int i = 0; i < MESH_BUFFER_DEPTH; i++){
        e_s = e_s && (x_queue[i] == state_2.x_queue[i]);
        e_s = e_s && (y_queue[i] == state_2.y_queue[i]);
        e_s = e_s && (z_queue[i] == state_2.z_queue[i]);
        e_s = e_s && (ref_queue[i] == state_2.ref_queue[i]);
    }

    return e_s;
}

void traverse_state::pretty_print(){
    std::cout << "\n\033[1;32mtraverse_state: \033[0m {\n";

    named_value(current_depth);
    named_value(current_indice);
    named_value(block_beginning_indice);
    named_array(x_queue,MESH_BUFFER_DEPTH);
    named_array(y_queue,MESH_BUFFER_DEPTH);
    named_array(z_queue,MESH_BUFFER_DEPTH);
    named_array(ref_queue,MESH_BUFFER_DEPTH);

    std::cout << "}\n";
}


bool traverse_state::is_ghost(physics_mesh &mesh){
  if(get_x() == 0 || get_y() == 0 || get_z() == 0
      || get_x() == mesh.mesh_sizes[current_depth]-1
      || get_y() == mesh.mesh_sizes[current_depth]-1
      || get_z() == mesh.mesh_sizes[current_depth]-1){
        return true;
  }
  else{
    return false;
  }
}


void traverse_state::cell_world_lookup(physics_mesh &mesh, float &x, float &y, float &z){
  x = 0;
  y = 0;
  z = 0;
  for(int i = 0; i < ((current_depth+1)) && (i < MESH_BUFFER_DEPTH); i++){
    x += mesh.world_scale[i]*(x_queue[i]-1); //ghost offset
    y += mesh.world_scale[i]*(y_queue[i]-1);
    z += mesh.world_scale[i]*(z_queue[i]-1);
  }
}

//
// int world_cell_index_lookup(float x, float y, float z, int max_depth){
//   int x_,y_,z_;
//   for(int depth = 0, depth < max_depth; depth++){
//       x_ = (int)floor(x/(mesh_scale[depth]));
//       y_ = (int)floor(y/(mesh_scale[depth]));
//       z_ = (int)floor(z/(mesh_scale[depth])); //remember ghosts!
//
//       x -= x_*mesh_scale[depth]
//   }
// }

void traverse_state::descend_into(physics_mesh &mesh, bool ignore_ghosts){
    ref_queue[current_depth] = block_beginning_indice;
    current_depth += 1;
    ref_queue[current_depth] = mesh.refined_indices[current_indice];
    block_beginning_indice = mesh.refined_indices[current_indice];
    set_x(ignore_ghosts);
    set_y(ignore_ghosts);
    set_z(ignore_ghosts);
    update_position(mesh);

}

void traverse_state::ascend_from(physics_mesh &mesh, bool ignore_ghosts){
    current_depth-=1;
    update_position(mesh);
}

void traverse_state::update_position(physics_mesh &mesh){
    block_beginning_indice = ref_queue[current_depth];

    current_indice = block_beginning_indice+
                    idx(get_x(),get_y(),get_z(),mesh.mesh_sizes[current_depth]);

}

bool physics_mesh::breadth_first(traverse_state &state, int start_depth, int end_depth, int ignore_ghosts){

    /*
    A traverse through all the cells of all the blocks at a specified level.
    'Breadth first' is perhaps a bit misleading; more precisely it's depth first then breadth across
    depth
    Iterates through the x,y,z values of a block,
    ref_queue stores block indices of parent blocks.

    g o o o o o... g
    ^
    block_beginning_indice

    */

    while(true){
        if(state.started_traverse){
            state.set_x(ignore_ghosts);
            state.set_y(ignore_ghosts);
            state.set_z(ignore_ghosts);
            state.started_traverse=false;
        }
        else{
            state.x_queue[state.current_depth]++;
        }

        //ensure that we don't start in the corner if ghosts are to be ignored.

        if(state.get_x() == (mesh_sizes[state.current_depth]-ignore_ghosts)) {state.set_x(ignore_ghosts); state.y_queue[state.current_depth]++;}
        if(state.get_y() == (mesh_sizes[state.current_depth]-ignore_ghosts)) {state.set_y(ignore_ghosts); state.z_queue[state.current_depth]++;}

        // if(state.z < mesh_sizes[state.current_depth]){
        //     state.x_queue[state.current_depth] = state.x;
        //     state.y_queue[state.current_depth] = state.y;
        //     state.z_queue[state.current_depth] = state.z;
        // }

        bool just_visited = 0;

        while(true){

            state.update_position(*this);

          if(state.current_depth < end_depth && refined_indices[state.current_indice]
                                && !just_visited && state.get_z() < mesh_sizes[state.current_depth]){
              //Descend
              state.descend_into(*this, ignore_ghosts);
              continue;
          }

          if(state.get_z() == (mesh_sizes[state.current_depth]-ignore_ghosts)){

              if(state.current_depth == 0){
                  return false;
              }
              state.set_x(0); //not strictly necessary, just cleans up a bit
              state.set_y(0);
              state.set_z(0);
              state.current_depth-=1;

              just_visited = true;
              continue;
          }

          break;
        }

        if(state.current_depth >= start_depth && state.current_depth <= end_depth){
            return true;
        }
        else{
            continue;
        }
    }
}


//avoids complexity in updating x,y,z values constantly
//though it does add one level of pointer indirection for every update,
//rather than just occasionally.
int traverse_state::get_x(){
    return x_queue[current_depth];
}
int traverse_state::get_y(){
    return y_queue[current_depth];
}
int traverse_state::get_z(){
    return z_queue[current_depth];
}
void traverse_state::set_x(int i){
    x_queue[current_depth] = i;
}
void traverse_state::set_y(int i){
    y_queue[current_depth] = i;
}
void traverse_state::set_z(int i){
    z_queue[current_depth] = i;
}
