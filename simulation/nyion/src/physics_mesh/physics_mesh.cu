#include "hip/hip_runtime.h"
#include "nyion.hpp"

#include "physics_mesh.hpp"

//most of this isn't cuda: we just add the .cu extension to get cuda to link some of the __device__ code.

//constructor
__host__ physics_mesh::physics_mesh(int (&set_mesh_sizes)[MESH_BUFFER_DEPTH], int init_mesh_depth){
    //set scales and sizes
    assert("Increase MESH_BUFFER_DEPTH" && MESH_BUFFER_DEPTH >= init_mesh_depth);

    for(int i = 0; i < init_mesh_depth; i++){ mesh_sizes[i] = set_mesh_sizes[i]; };
    for(int i = init_mesh_depth; i < MESH_BUFFER_DEPTH; i++){ mesh_sizes[i] = 3; };

    //initialize root on unrolled array
    block_depth_lookup[0] = 0;
    for(int i = 1; i < MESH_BUFFER_DEPTH+1; i++){ block_depth_lookup[i] = 1; };


    compute_world_scale();

    //on construction, initialize root
    buffer_end_pointer = cube(mesh_sizes[0]);

    //and allocate memory
    temperature = new float[MESH_BUFFER_SIZE];
    potential = new float[MESH_BUFFER_SIZE];
    space_charge = new int32_t[MESH_BUFFER_SIZE];
    boundary_conditions = new uint16_t[MESH_BUFFER_SIZE];
    refined_indices = new uint32_t[MESH_BUFFER_SIZE];
    ghost_linkages = new uint32_t[MESH_BUFFER_SIZE];
    block_indices = new uint32_t[MESH_BUFFER_SIZE];//max blocks?

    //std::fill not available on GPU.
    for(int i = 0; i < MESH_BUFFER_SIZE; i++){
        temperature[i] = 0;
        potential[i] = 0;
        space_charge[i] = 0; //a canary (perhaps -inf?) might be useful
        boundary_conditions[i] = 0;
        refined_indices[i] = 0;
        ghost_linkages[i] = 0;
        block_indices[i] = 0;
    }
}

__device__ __host__ int physics_mesh::blocks_on_level(int depth){
    return block_depth_lookup[depth+1]-block_depth_lookup[depth];
}


__device__ __host__ void physics_mesh::block_list_insert(int depth, int refined_indice){
    //to accomodate iterating over blocks without traversing a tree,
    //block IDs are also stored in an array.
    //block_num stores how many indices are in each level.
    // we don't actually care about the order of block_indices
    // between levels: popping can be a quick search.
    // see digraph.
    // Having the block_depth_lookup accumulative
    // prevents us from having to sum on hot-loop operations,
    // at a cost of needing one more indice to

    int tail_position = block_depth_lookup[depth];

    //number after - to shift minimum possible
    int end_position = block_depth_lookup[MESH_BUFFER_DEPTH-1];

    //shift data up
    for(int i = end_position; i > tail_position; i--){
        block_indices[i] = block_indices[i-1];
    }

    block_indices[tail_position] = refined_indice;

    for(int i = depth+1; i < MESH_BUFFER_DEPTH+1; i++){ block_depth_lookup[i]+=1; }

}

__device__ __host__ void physics_mesh::refine_cell(int current_depth, int current_indice){
    //this will be called from depth 0:...
    //the refinement will be added to depth 1:...

    if(refined_indices[current_indice]){ //if mesh is already refined, ignore.
        return;
    }

    assert("Tried to refine too deep!" && current_depth+1 < MESH_BUFFER_DEPTH);

    refined_indices[current_indice] = buffer_end_pointer;

    block_list_insert(current_depth+1, buffer_end_pointer);

    buffer_end_pointer += cube(mesh_sizes[current_depth+1]);

    compute_world_scale();
}


__device__ __host__ void physics_mesh::compute_world_scale(){
    //we want to quickly init mesh_sizes like {3,3,5} for testing.
    //however, that
    for(int i = 0; i < MESH_BUFFER_DEPTH; i++){ world_scale[i] = 0; };
    // pre-compute scales
    float scale = ROOT_WORLD_SCALE;
    for(int i = 0; i < MESH_BUFFER_DEPTH; i++){
        assert("Mesh size must be > 2" && mesh_sizes[i]-2 > 0);
        assert("Mesh size must be < 200" && mesh_sizes[i]-2 < 200);
        scale /= mesh_sizes[i]-2; //-2 compensates for ghost points.
        world_scale[i] = scale;
    } // TODO: Scales must be re-computed if the size changes!
}


void physics_mesh::set_level_ghost_linkages(){

}


template <class T>
void add_to_object(json &object, T * input, std::string name, int n){
        std::vector<float> temp;
        temp.assign(input, input+n);
        object[name] = temp;
}

__host__ json physics_mesh::serialize(){
    json object;

    add_to_object(object, world_scale, "world_scale", MESH_BUFFER_DEPTH);
    add_to_object(object, mesh_sizes, "mesh_sizes", MESH_BUFFER_DEPTH);
    add_to_object(object, block_depth_lookup, "block_depth_lookup", MESH_BUFFER_DEPTH+1);

    object["buffer_end_pointer"] = buffer_end_pointer;

    add_to_object(object, temperature, "temperature", buffer_end_pointer);
    add_to_object(object, potential, "potential", buffer_end_pointer);
    add_to_object(object, space_charge, "space_charge", buffer_end_pointer);
    add_to_object(object, boundary_conditions, "boundary_conditions", buffer_end_pointer);
    add_to_object(object, refined_indices, "refined_indices", buffer_end_pointer);
    add_to_object(object, ghost_linkages, "ghost_linkages", buffer_end_pointer);
    add_to_object(object, block_indices, "block_indices", buffer_end_pointer);



    return object;
}




#define IS_EQUAL_MACRO(NAME) is_equal = is_equal && (NAME[i] == mesh_2.NAME[i]);

__host__ bool physics_mesh::equals(physics_mesh &mesh_2){
    bool is_equal = true;

    is_equal = is_equal && (buffer_end_pointer == mesh_2.buffer_end_pointer);

    for(int i = 0; i < MESH_BUFFER_SIZE; i++){
        IS_EQUAL_MACRO(temperature);
        IS_EQUAL_MACRO(potential);
        IS_EQUAL_MACRO(space_charge);
        IS_EQUAL_MACRO(boundary_conditions);
        IS_EQUAL_MACRO(refined_indices);
        IS_EQUAL_MACRO(ghost_linkages);
        IS_EQUAL_MACRO(block_indices);
    }

    for(int i = 0; i < MESH_BUFFER_DEPTH; i++){
        is_equal = is_equal && (mesh_sizes[i] == mesh_2.mesh_sizes[i]);
        is_equal = is_equal && (world_scale[i] == mesh_2.world_scale[i]);

    }

    for(int i = 0; i < MESH_BUFFER_DEPTH+1; i++){
        is_equal = is_equal && (block_depth_lookup[i] == mesh_2.block_depth_lookup[i]);
    }

    return is_equal;
}




__host__ void physics_mesh::pretty_print(){
    std::cout << "\n\033[1;32mphysics_mesh: \033[0m {\n";

    named_array(world_scale, MESH_BUFFER_DEPTH);
    named_array(mesh_sizes, MESH_BUFFER_DEPTH);
    named_array(temperature, buffer_end_pointer);
    named_array(potential, buffer_end_pointer);
    named_array(space_charge, buffer_end_pointer);
    named_array(boundary_conditions, buffer_end_pointer);
    named_array(refined_indices, buffer_end_pointer);
    named_array(ghost_linkages, buffer_end_pointer);
    named_array(block_indices, buffer_end_pointer);
    named_array(block_depth_lookup, MESH_BUFFER_DEPTH+1);

    std::cout << "}\n";
}


//destructor
physics_mesh::~physics_mesh(){
    //on destruction,
    delete [] temperature;
    delete [] potential;
    delete [] space_charge;
    delete [] boundary_conditions;
    delete [] refined_indices;
    delete [] ghost_linkages;
    delete [] block_indices;
}


int idx(int x, int y, int z, int len){
  return (x + (y*len) + (z*len*len));
}
