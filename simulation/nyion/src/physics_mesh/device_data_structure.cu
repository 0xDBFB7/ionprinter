#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"


// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
__global__ void device_copy_ghost_values_kernel(const physics_mesh &device_struct, float ** values, int depth) {

    int direction = threadIdx.x;

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + transform_idx(device_struct.mesh_sizes[depth]-1,threadIdx.y,threadIdx.z,
                                                            device_struct.mesh_sizes[depth], direction);

    if(device_struct.ghost_linkages[this_cell]){
        int value_from_indice = device_struct.ghost_linkages[this_cell];
        (*values)[this_cell] = (*values)[value_from_indice];
    }

}

// template <class T>
void physics_mesh::device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.

    int num_blocks = (*host_struct).blocks_on_level(depth);
    dim3 threads_per_block(6, (*host_struct).mesh_sizes[depth], (*host_struct).mesh_sizes[depth]);
    //
    device_copy_ghost_values_kernel<<<num_blocks, threads_per_block>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
    //error checking should be done externally; we don't want to hipDeviceSynchronize after every operation!
}
