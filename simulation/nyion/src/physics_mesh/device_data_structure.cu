#include "hip/hip_runtime.h"
#include "device_data_structure.hpp"

const int max_cells_per_cuda_block = 1024;



//GPU supports 2^n*65535*65535 blocks of 1024 threads each.
//it is assumed that one yz plane of each block cannot exceed 1024 cells;
//that is, mesh_sizes cannot be > 32.
void set_GPU_dimensions(physics_mesh * host_struct, dim3 &blocks, dim3 &threads, int depth){
    //number of refined blocks actually present in mesh on this level
    int num_physical_blocks = (*host_struct).blocks_on_level(depth);
    //physical blocks on this level are all the same size,
    int physical_block_width = (*host_struct).mesh_sizes[depth]-2; //ghosts!

    int sub_blocks = ceil(((float)cube(physical_block_width))/((float)max_cells_per_cuda_block));
    while(physical_block_width % sub_blocks != 0){
        //if the block can't be divided evenly in Z
        sub_blocks++;
        //will get to physical_block_width at most.
    }

    blocks.PHYSICAL_BLOCKS = num_physical_blocks;
    blocks.SUB_BLOCKS = sub_blocks;

    threads.x = physical_block_width;
    threads.y = physical_block_width;
    threads.z = physical_block_width/sub_blocks;
}


//+1 to ignore ghosts
// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
__global__ void device_copy_ghost_values_kernel(physics_mesh &device_struct, float ** values, int depth) {

    int direction = threadIdx.x;

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.PHYSICAL_BLOCKS];

    int x = device_struct.mesh_sizes[depth]-1;
    int y = (threadIdx.y)+1;
    int z = ((((device_struct.mesh_sizes[depth]-2)/gridDim.SUB_BLOCKS)*blockIdx.SUB_BLOCKS)+threadIdx.z)+1;

    int this_cell = this_block + transform_idx(x,y,z, device_struct.mesh_sizes[depth], direction);

    if(device_struct.ghost_linkages[this_cell]){
        int value_from_indice = device_struct.ghost_linkages[this_cell];
        (*values)[this_cell] = (*values)[value_from_indice];
    }
}

// template <class T>
void physics_mesh::device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    if(depth == 0) return;
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.
    dim3 threads;
    dim3 blocks;

    set_GPU_dimensions(host_struct,blocks,threads,depth);
    threads.x = 6; //override

    device_copy_ghost_values_kernel<<<blocks, threads>>>(*device_struct, values, depth);
    // gpu_error_check( hipPeekAtLastError() );
    // gpu_error_check( hipDeviceSynchronize() );
    //error checking now handled externally.
}


//+1 to ignore ghosts
__global__ void device_jacobi_relax_kernel(physics_mesh &device_struct, float ** input, float * output, int depth) {
    //see https://people.eecs.berkeley.edu/~demmel/cs267/lecture24/lecture24.html

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int x = threadIdx.x+1;
    int y = threadIdx.y+1; //if there's a bug here, make sure gridDim and blockDim are correct.
    int z = ((((device_struct.mesh_sizes[depth]-2)/gridDim.SUB_BLOCKS)*blockIdx.SUB_BLOCKS)+threadIdx.z)+1;

    int this_cell = this_block + idx(x, y, z, device_struct.mesh_sizes[depth]);

    bool boundary_condition_check = (device_struct.boundary_conditions[this_cell] == 0); //1 if cell should be updated

    float sum = 0;
    sum += (*input)[this_block + idx(x+1, y, z, device_struct.mesh_sizes[depth])];
    sum += (*input)[this_block + idx(x-1, y, z, device_struct.mesh_sizes[depth])];
    sum += (*input)[this_block + idx(x, y+1, z, device_struct.mesh_sizes[depth])];
    sum += (*input)[this_block + idx(x, y-1, z, device_struct.mesh_sizes[depth])];
    sum += (*input)[this_block + idx(x, y, z+1, device_struct.mesh_sizes[depth])];
    sum += (*input)[this_block + idx(x, y, z-1, device_struct.mesh_sizes[depth])];
    sum += (device_struct.space_charge[this_cell]/EPSILON_ZERO);

    (output)[this_cell] = ((1-boundary_condition_check)*(*input)[this_cell]) //leave cell unchanged if a boundary is present - saves a jump
                                    + (boundary_condition_check * (sum)/(6.0f));

}

__global__ void device_copy_kernel(physics_mesh &device_struct, float * input, float ** output, int depth) {
    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int x = threadIdx.x+1;
    int y = threadIdx.y+1; //if there's a bug here, make sure gridDim and blockDim are correct.
    int z = ((((device_struct.mesh_sizes[depth]-2)/gridDim.SUB_BLOCKS)*blockIdx.SUB_BLOCKS)+threadIdx.z)+1;

    int this_cell = this_block + idx(x, y, z, device_struct.mesh_sizes[depth]);

    (*output)[this_cell] = (input)[this_cell];
}


void physics_mesh::device_jacobi_relax(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int iterations, int depth){
    dim3 threads;
    dim3 blocks;

    set_GPU_dimensions(host_struct,blocks,threads,depth);

    float * device_temp;
    //this is basically free since nothing needs to be zeroed
    gpu_error_check(hipMalloc(&device_temp, sizeof(float)*(*host_struct).buffer_end_pointer));

    physics_mesh::device_copy_ghost_values(host_struct, device_struct, values, depth);

    for(int i = 0; i < iterations; i++){
        device_jacobi_relax_kernel<<<blocks, threads>>>(*device_struct, values, device_temp, depth);

        device_copy_kernel<<<blocks, threads>>>(*device_struct, device_temp, values, depth); //double-buffering adds quite a bit of complexity.

        device_copy_ghost_values(host_struct, device_struct, values, depth);

    }


    //also basically free
    hipFree(device_temp);

    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );


}
