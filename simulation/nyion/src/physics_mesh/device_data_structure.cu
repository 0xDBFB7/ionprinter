#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"

const int max_cells_per_cuda_block = 1024;



//GPU supports 2^n*65535*65535 blocks of 1024 threads each.
//it is assumed that one yz plane of each block cannot exceed 1024 cells;
//that is, mesh_sizes cannot be > 32.
void set_GPU_dimensions(physics_mesh * host_struct, dim3 &blocks, dim3 &threads, int depth){
    //number of refined blocks actually present in mesh on this level
    int num_physical_blocks = (*host_struct).blocks_on_level(depth);
    //physical blocks on this level are all the same size,
    int physical_block_width = (*host_struct).mesh_sizes[depth]-2; //ghosts!

    int sub_blocks = ceil(((float)cube(physical_block_width))/((float)max_cells_per_cuda_block));
    while(physical_block_width % sub_blocks != 0){
        //if the block can't be divided evenly in Z
        sub_blocks++;
    }

    blocks.PHYSICAL_BLOCKS = num_physical_blocks;
    blocks.SUB_BLOCKS = sub_blocks;

    threads.x = physical_block_width;
    threads.y = physical_block_width;
    threads.z = physical_block_width/sub_blocks;
}


//+1 to ignore ghosts
// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
__global__ void device_copy_ghost_values_kernel(physics_mesh &device_struct, float ** values, int depth) {

    int direction = threadIdx.x;

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.PHYSICAL_BLOCKS];

    int x = device_struct.mesh_sizes[depth]-1;
    int y = (threadIdx.y)+1;
    int z = ((((device_struct.mesh_sizes[depth]-2)/blockDim.SUB_BLOCKS)*blockIdx.SUB_BLOCKS)+threadIdx.z)+1;

    int this_cell = this_block + transform_idx(x,y,z, device_struct.mesh_sizes[depth], direction);

    if(device_struct.ghost_linkages[this_cell]){
        int value_from_indice = device_struct.ghost_linkages[this_cell];
        (*values)[this_cell] = (*values)[value_from_indice];
    }
}

// template <class T>
void physics_mesh::device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.
    dim3 threads;
    dim3 blocks;

    set_GPU_dimensions(host_struct,blocks,threads,depth);
    threads.x = 6; //override

    device_copy_ghost_values_kernel<<<blocks, threads>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
    // error checking should be done externally; we don't want to hipDeviceSynchronize after every operation!
}


//+1 to ignore ghosts
__global__ void device_jacobi_relax_kernel(physics_mesh &device_struct, float ** values, int depth) {

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + idx(threadIdx.x+1,threadIdx.y+1,threadIdx.z+1, device_struct.mesh_sizes[depth]);

    int value_from_indice = device_struct.ghost_linkages[this_cell];
    (*values)[this_cell] = (*values)[value_from_indice];
}


void physics_mesh::device_jacobi_relax(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    int num_blocks = (*host_struct).blocks_on_level(depth);
    dim3 threads_per_block((*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2); //ignore ghosts

    device_jacobi_relax_kernel<<<num_blocks, threads_per_block>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
}
