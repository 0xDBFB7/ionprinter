#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"


// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
__global__ void device_copy_ghost_values_kernel(const physics_mesh &device_mesh, float * values, int depth) {

    int direction = threadIdx.x;

    int this_block = block_indices[device_mesh.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + transform_idx(device_mesh.mesh_sizes[depth]-1,threadIdx.y,threadIdx.z,
                                                            device_mesh.mesh_sizes[depth], direction);

    if(ghost_linkages[this_cell]){
        int value_from_indice = ghost_linkages[this_cell]);
        values[this_cell] = values[value_from_indice];
    }

}


void device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, int level){
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.

    int num_blocks = (*host_struct).blocks_on_level(level);
    dim3 threads_per_block(6, (*host_struct).mesh_sizes[level], (*host_struct).mesh_sizes[level]);

    device_copy_ghost_values_kernel<<<num_blocks, threads_per_block>>>(*device_struct);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
}
