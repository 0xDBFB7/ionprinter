#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"

const int max_cells_per_cuda_block = 1024;

//GPU supports 2^n*65535*65535 blocks of 1024 threads each.
//the mesh exceeds this size significantly.
//so each physical block is broken down further.
//We will assume that a single row of mesh_sizes can never be > 1024
//num_threads_x
void set_GPU_dimensions(physics_mesh * host_struct, dim3 &blocks, dim3 &threads, int depth){
    //number of refined blocks actually present in mesh on this level
    int num_physical_blocks = (*host_struct).blocks_on_level(depth);
    //physical blocks on this level are all the same size,
    int physical_block_width = (*host_struct).mesh_sizes[depth]-2; //ghosts!

    int sub_blocks = max_cells_per_cuda_block/physical_block_width;

    int sub_blocks_y = sub_blocks;
    int sub_blocks_z = 1; //?

    blocks.PHYSICAL_BLOCKS = num_physical_blocks;
    blocks.SUB_BLOCKS_Y = sub_blocks_y;
    blocks.SUB_BLOCKS_Z = sub_blocks_z;

    threads.x = physical_block_width;
    threads.y = physical_block_width/sub_blocks_y;
    threads.z = physical_block_width/sub_blocks_z;

}


//+1 to ignore ghosts
// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
//blockIdx.x is the number of physical blocks in this mesh
//blockIdx y,z are the number of 'imaginary' sub-blocks to fulfil thread requirements
//threadIdx.x is the facet of the cube currently being operated on
//threadIdx y,z are the positions on the facet.
__global__ void device_copy_ghost_values_kernel(physics_mesh &device_struct, float ** values, int depth) {

    int direction = threadIdx.x;

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int y = ((((device_struct.mesh_sizes[depth]-2)/blockDim.y)*blockIdx.y)+threadIdx.y)+1;
    int z = ((((device_struct.mesh_sizes[depth]-2)/blockDim.z)*blockIdx.z)+threadIdx.z)+1;

    int this_cell = this_block + transform_idx(device_struct.mesh_sizes[depth]-1,y,z,
                                                            device_struct.mesh_sizes[depth], direction);

    if(device_struct.ghost_linkages[this_cell]){
        int value_from_indice = device_struct.ghost_linkages[this_cell];
        (*values)[this_cell] = (*values)[value_from_indice];
    }
}

// template <class T>
void physics_mesh::device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.
    dim3 threads;
    dim3 blocks;
    //
    // set_GPU_dimensions(host_struct,blocks,threads,depth);
    // threads.x = 6; //override
    //
    // device_copy_ghost_values_kernel<<<blocks, threads>>>(*device_struct, values, depth);
    // gpu_error_check( hipPeekAtLastError() );
    // gpu_error_check( hipDeviceSynchronize() );
    //error checking should be done externally; we don't want to hipDeviceSynchronize after every operation!
}


//+1 to ignore ghosts
__global__ void device_jacobi_relax_kernel(physics_mesh &device_struct, float ** values, int depth) {

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + idx(threadIdx.x+1,threadIdx.y+1,threadIdx.z+1, device_struct.mesh_sizes[depth]);

    int value_from_indice = device_struct.ghost_linkages[this_cell];
    (*values)[this_cell] = (*values)[value_from_indice];
}


void physics_mesh::device_jacobi_relax(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    int num_blocks = (*host_struct).blocks_on_level(depth);
    dim3 threads_per_block((*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2); //ignore ghosts

    device_jacobi_relax_kernel<<<num_blocks, threads_per_block>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
}
