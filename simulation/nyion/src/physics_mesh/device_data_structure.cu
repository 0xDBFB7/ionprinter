#include "hip/hip_runtime.h"
#include "physics_mesh.hpp"

//+1 to ignore ghosts
// a simple if(ghost_linkages[cell]) over all cells might end up being faster.
__global__ void device_copy_ghost_values_kernel(const physics_mesh &device_struct, float ** values, int depth) {

    int direction = threadIdx.x;

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + transform_idx(device_struct.mesh_sizes[depth]-1,threadIdx.y+1,threadIdx.z+1,
                                                            device_struct.mesh_sizes[depth], direction);

    if(device_struct.ghost_linkages[this_cell]){
        int value_from_indice = device_struct.ghost_linkages[this_cell];
        (*values)[this_cell] = (*values)[value_from_indice];
    }

}

// template <class T>
void physics_mesh::device_copy_ghost_values(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    //here we're using the third dimension as the 'cube facet count'
    //since iterating over facets only requires two dimensions:
    //X and Y.
    //level must be >0.
    //assumes that parameters of the mesh are consistent from host to device.
    //This may not be a safe assumption.

    int num_blocks = (*host_struct).blocks_on_level(depth);
    dim3 threads_per_block(6, (*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2); //ignore ghosts
    //
    device_copy_ghost_values_kernel<<<num_blocks, threads_per_block>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
    //error checking should be done externally; we don't want to hipDeviceSynchronize after every operation!
}


//+1 to ignore ghosts
__global__ void device_jacobi_relax_kernel(const physics_mesh &device_struct, float ** values, int depth) {

    int this_block = device_struct.block_indices[device_struct.block_depth_lookup[depth]+blockIdx.x];

    int this_cell = this_block + idx(threadIdx.x+1,threadIdx.y+1,threadIdx.z+1, device_struct.mesh_sizes[depth]);

    int value_from_indice = device_struct.ghost_linkages[this_cell];
    (*values)[this_cell] = (*values)[value_from_indice];
}


void physics_mesh::device_jacobi_relax(physics_mesh * host_struct, physics_mesh * device_struct, float ** values, int depth){
    int num_blocks = (*host_struct).blocks_on_level(depth);
    dim3 threads_per_block(6, (*host_struct).mesh_sizes[depth]-2, (*host_struct).mesh_sizes[depth]-2); //ignore ghosts

    device_jacobi_relax_kernel<<<num_blocks, threads_per_block>>>(*device_struct, values, depth);
    gpu_error_check( hipPeekAtLastError() );
    gpu_error_check( hipDeviceSynchronize() );
}
