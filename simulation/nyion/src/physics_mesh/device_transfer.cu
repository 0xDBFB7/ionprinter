/*

Having a tree relationship between with levels and blocks makes some kind of sense.
Everything multigrid is naturally tree-based,
having a structure to traverse is great for cell-world and world-cell lookups,
and to generate the linkages between ghosts.
It's also easier to code the construction of meshes.

However, traversing the tree takes jumps and is generally poorly optimized,
especially on a GPU, where a jump stalls the warp.
Once the trees are established on the heap, however, a simple list of indices
can be constructed to traverse linearly.

It might be possible to do away with the tree entirely, but
trying to visualize 'neighbors' with arbitrarily placed blocks
does my head in.

It's important to note that this entire implementation completely disregards
the cache altogether. The indice system means the next block could be halfway to Manitoba
out of cache; there's no cache concurrency, congruency, or congealency; the
&mesh argument means we're probably bringing 20x as much data along as we need for each function;
etc.

If one were truly smart, one could probably eek the same performance out of a
homogenous computing system. I am not.

//block_list
//block_sizes
//connections

//
//Ryzen 7: 0.25 TF/40 gbps. https://en.wikichip.org/wiki/amd/ryzen_7/1700
//GTX 1060: 3 TFlops/160 GBps.
//RTX 2070 Super: 8.2 TF/448.0 GBps - 21 to 51 TFlops half precision.
//V100 hits 14TF/900 GBps + 100TF half.
//https://en.wikipedia.org/wiki/List_of_Nvidia_graphics_processing_units
//
//Special thanks to http://ianfinlayson.net/class/cpsc425/notes/cuda-random!
//
*/

//if all the pointers were put to the end of physics_mesh, perhaps we could simplify the pointer wipe code?
//hipMemcpy(...,&(struct) -  &(struct->canary)...)
//
// oh, no, structure order doesn't seem to be preserved in some cases.

#include "physics_mesh.hpp"
#include "struct_transfer_helpers.hpp"

void physics_mesh::device_constructor(physics_mesh ** device_struct){
    //construct the struct itself
    gpu_error_check(hipMalloc(device_struct, sizeof(physics_mesh)));
    //zero the struct memory - just in case
    gpu_error_check(hipMemset((*device_struct),0,sizeof(physics_mesh)));

    //construct the arrays and copy to the GPU
    constructificate_array_on_struct(&((**device_struct).temperature), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).potential), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).space_charge), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).boundary_conditions), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).refined_indices), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).ghost_linkages), MESH_BUFFER_SIZE);
    constructificate_array_on_struct(&((**device_struct).block_indices), MESH_BUFFER_SIZE);

}

void physics_mesh::copy_to_device(physics_mesh ** device_struct, physics_mesh ** host_struct){
    //double pointer required to preserve malloc edit
    uint32_t length = (**host_struct).buffer_end_pointer; //length from host

    //copy array and save a copy of each array pointer for later
    COPY_ARRAY_TO_DEVICE(float, temperature, length);
    COPY_ARRAY_TO_DEVICE(float, potential, length);
    COPY_ARRAY_TO_DEVICE(int32_t, space_charge, length);
    COPY_ARRAY_TO_DEVICE(uint16_t, boundary_conditions, length);
    COPY_ARRAY_TO_DEVICE(uint32_t, refined_indices, length);
    COPY_ARRAY_TO_DEVICE(uint32_t, ghost_linkages, length);
    COPY_ARRAY_TO_DEVICE(uint32_t, block_indices, length);


    //copy struct itself, wiping all the pointers,
    gpu_error_check(hipMemcpy(*device_struct, *host_struct, sizeof(physics_mesh), hipMemcpyHostToDevice));

    //then re-copy the pointers.
    COPY_POINTERS_BACK_TO_DEVICE(float, temperature, length);
    COPY_POINTERS_BACK_TO_DEVICE(float, potential, length);
    COPY_POINTERS_BACK_TO_DEVICE(int32_t, space_charge, length);
    COPY_POINTERS_BACK_TO_DEVICE(uint16_t, boundary_conditions, length);
    COPY_POINTERS_BACK_TO_DEVICE(uint32_t, refined_indices, length);
    COPY_POINTERS_BACK_TO_DEVICE(uint32_t, ghost_linkages, length);
    COPY_POINTERS_BACK_TO_DEVICE(uint32_t, block_indices, length);

}
//There's a PCIe latency thing here, since we're going * -> host, data -> device,
//but whatever!


void physics_mesh::copy_to_host(physics_mesh ** device_struct, physics_mesh ** host_struct){

    //save host array pointers for after the wipe
    SAVE_HOST_ARRAY_POINTER(float, temperature, length);
    SAVE_HOST_ARRAY_POINTER(float, potential, length);
    SAVE_HOST_ARRAY_POINTER(int32_t, space_charge, length);
    SAVE_HOST_ARRAY_POINTER(uint16_t, boundary_conditions, length);
    SAVE_HOST_ARRAY_POINTER(uint32_t, refined_indices, length);
    SAVE_HOST_ARRAY_POINTER(uint32_t, ghost_linkages, length);
    SAVE_HOST_ARRAY_POINTER(uint32_t, block_indices, length);

    //copy struct itself, wiping all the host pointers to arrays,
    gpu_error_check(hipMemcpy(*host_struct, *device_struct, sizeof(physics_mesh), hipMemcpyDeviceToHost));

    //then restore host pointers to host arrays

    RESTORE_HOST_ARRAY_POINTER(float, temperature, length);
    RESTORE_HOST_ARRAY_POINTER(float, potential, length);
    RESTORE_HOST_ARRAY_POINTER(int32_t, space_charge, length);
    RESTORE_HOST_ARRAY_POINTER(uint16_t, boundary_conditions, length);
    RESTORE_HOST_ARRAY_POINTER(uint32_t, refined_indices, length);
    RESTORE_HOST_ARRAY_POINTER(uint32_t, ghost_linkages, length);
    RESTORE_HOST_ARRAY_POINTER(uint32_t, block_indices, length);

    //length from device was already copied to host via struct
    uint32_t length = ((**host_struct).buffer_end_pointer);

    COPY_ARRAY_TO_HOST(float, temperature, length);
    COPY_ARRAY_TO_HOST(float, potential, length);
    COPY_ARRAY_TO_HOST(int32_t, space_charge, length);
    COPY_ARRAY_TO_HOST(uint16_t, boundary_conditions, length);
    COPY_ARRAY_TO_HOST(uint32_t, refined_indices, length);
    COPY_ARRAY_TO_HOST(uint32_t, ghost_linkages, length);
    COPY_ARRAY_TO_HOST(uint32_t, block_indices, length);

}

void physics_mesh::device_destructor(physics_mesh ** device_struct){

    defenestrate_array_from_struct(&((**device_struct).temperature));
    defenestrate_array_from_struct(&((**device_struct).potential));
    defenestrate_array_from_struct(&((**device_struct).space_charge));
    defenestrate_array_from_struct(&((**device_struct).boundary_conditions));
    defenestrate_array_from_struct(&((**device_struct).refined_indices));
    defenestrate_array_from_struct(&((**device_struct).ghost_linkages));
    defenestrate_array_from_struct(&((**device_struct).block_indices));

    //then destroy the struct itself.
    gpu_error_check(hipFree(&(**device_struct)));
}
